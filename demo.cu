#include <iostream>
#include <hip/hip_runtime.h>

#define N 1024  // 定义矩阵维度

// CUDA 核函数：矩阵乘法
__global__ void matrixMulKernel(float *A, float *B, float *C, int width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < width && col < width) {
        float value = 0.0f;
        for (int k = 0; k < width; ++k) {
            value += A[row * width + k] * B[k * width + col];
        }
        C[row * width + col] = value;
    }
}

// 辅助函数：初始化矩阵
void initializeMatrix(float *matrix, int width) {
    for (int i = 0; i < width * width; ++i) {
        matrix[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

// 辅助函数：打印矩阵
void printMatrix(const float *matrix, int width) {
    for (int i = 0; i < width; ++i) {
        for (int j = 0; j < width; ++j) {
            std::cout << matrix[i * width + j] << " ";
        }
        std::cout << std::endl;
    }
}

int main() {
    int matrixSize = N * N * sizeof(float);
    
    // 分配主机内存
    float *h_A = (float *)malloc(matrixSize);
    float *h_B = (float *)malloc(matrixSize);
    float *h_C = (float *)malloc(matrixSize);
    
    // 初始化主机矩阵
    initializeMatrix(h_A, N);
    initializeMatrix(h_B, N);
    
    // 分配设备内存
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, matrixSize);
    hipMalloc((void **)&d_B, matrixSize);
    hipMalloc((void **)&d_C, matrixSize);
    
    // 将数据从主机传输到设备
    hipMemcpy(d_A, h_A, matrixSize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, matrixSize, hipMemcpyHostToDevice);
    
    // 设置执行配置
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    // 启动 CUDA 核函数
    matrixMulKernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);
    
    // 将结果从设备传输回主机
    hipMemcpy(h_C, d_C, matrixSize, hipMemcpyDeviceToHost);
    
    // 打印结果矩阵
    // printMatrix(h_C, N); // 如需查看结果，可取消注释此行

    // 释放设备内存
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    // 释放主机内存
    free(h_A);
    free(h_B);
    free(h_C);
    
    std::cout << "矩阵乘法已完成。" << std::endl;
    
    return 0;
}
