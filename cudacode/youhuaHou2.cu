#include <iostream>
#include <hip/hip_runtime.h>

#define N 1024          // 定义矩阵维度
#define TILE_WIDTH 16   // 定义块大小

// CUDA 核函数：优化后的矩阵乘法，使用共享内存和循环展开
__global__ void matrixMulKernelOptimized(float *A, float *B, float *C, int width) {
    __shared__ float ds_A[TILE_WIDTH][TILE_WIDTH];  
    __shared__ float ds_B[TILE_WIDTH][TILE_WIDTH];
    int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int col = blockIdx.x * TILE_WIDTH + threadIdx.x;

    float value = 0.0f;

    // 分块计算
    for (int m = 0; m < (width + TILE_WIDTH - 1) / TILE_WIDTH; ++m) {
        // 加载A矩阵块到共享内存
        if (row < width && (m * TILE_WIDTH + threadIdx.x) < width)
            ds_A[threadIdx.y][threadIdx.x] = A[row * width + m * TILE_WIDTH + threadIdx.x];
        else
            ds_A[threadIdx.y][threadIdx.x] = 0.0f;

        // 加载B矩阵块到共享内存
        if ((m * TILE_WIDTH + threadIdx.y) < width && col < width)
            ds_B[threadIdx.y][threadIdx.x] = B[(m * TILE_WIDTH + threadIdx.y) * width + col];
        else
            ds_B[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();  // 等待所有线程完成数据加载

        // 计算乘积并累加，使用循环展开优化
        #pragma unroll
        for (int k = 0; k < TILE_WIDTH; ++k) {
            value += ds_A[threadIdx.y][k] * ds_B[k][threadIdx.x];
        }

        __syncthreads();  // 等待所有线程完成计算
    }

    // 将结果写回全局内存
    if (row < width && col < width)
        C[row * width + col] = value;
}

// 辅助函数：初始化矩阵
void initializeMatrix(float *matrix, int width) {
    for (int i = 0; i < width * width; ++i) {
        matrix[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

// 辅助函数：打印矩阵
void printMatrix(const float *matrix, int width) {
    for (int i = 0; i < width; ++i) {
        for (int j = 0; j < width; ++j) {
            std::cout << matrix[i * width + j] << " ";
        }
        std::cout << std::endl;
    }
}

int main() {
    int matrixSize = N * N * sizeof(float);

    // 分配主机内存
    float *h_A = (float *)malloc(matrixSize);
    float *h_B = (float *)malloc(matrixSize);
    float *h_C = (float *)malloc(matrixSize);

    // 初始化主机矩阵
    initializeMatrix(h_A, N);
    initializeMatrix(h_B, N);

    // 分配设备内存
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, matrixSize);
    hipMalloc((void **)&d_B, matrixSize);
    hipMalloc((void **)&d_C, matrixSize);

    // 将数据从主机传输到设备
    hipMemcpy(d_A, h_A, matrixSize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, matrixSize, hipMemcpyHostToDevice);

    // 设置执行配置
    dim3 threadsPerBlock(TILE_WIDTH, TILE_WIDTH);
    dim3 numBlocks((N + TILE_WIDTH - 1) / TILE_WIDTH,
                   (N + TILE_WIDTH - 1) / TILE_WIDTH);

    // 启动优化后的 CUDA 核函数
    matrixMulKernelOptimized<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();  // 等待所有线程完成

    // 将结果从设备传输回主机
    hipMemcpy(h_C, d_C, matrixSize, hipMemcpyDeviceToHost);

    // 打印结果矩阵（如需查看，请取消以下注释）
    // printMatrix(h_C, N);

    // 释放设备内存
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // 释放主机内存
    free(h_A);
    free(h_B);
    free(h_C);

    std::cout << "优化后的矩阵乘法已完成。" << std::endl;

    return 0;
}