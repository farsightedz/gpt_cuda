#include <hip/hip_runtime.h>
#include <iostream>

__global__ void efficientKernel(float *data, int N) {
    
    float a = 0, b = 0, c = 0, d = 0;
    float e = 0, f = 0, g = 0, h = 0;
    float i = 0, j = 0, k = 0, l = 0;
    float m = 0, n = 0, o = 0, p = 0;
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) {
        
        a = b + c; d = e + f;
        g = h + i; j = k + l;
        m = n + o; p = a + b;

        data[idx] = p;
    }
}

int main() {
    const int N = 1 << 20;  
    const int threadsPerBlock = 512; 
    const int numBlocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    float *d_data;
    size_t size = N * sizeof(float);

    // 分配设备内存
    hipMalloc(&d_data, size);

    // 初始化数据
    efficientKernel<<<numBlocks, threadsPerBlock>>>(d_data, N);

    // 同步设备，检查错误
    hipDeviceSynchronize();

    // 释放设备内存
    hipFree(d_data);

    return 0;
}
