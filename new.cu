#include "hip/hip_runtime.h"
__global__ void matrixMulKernel(float *A, float *B, float *C, int width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < width && col < width) {
        float value = 0.0f;
        for (int k = 0; k < width; ++k) {
            value += A[row * width + k] * B[k * width + col];
        }
        C[row * width + col] = value;
    }
}
\n
__global__ void matrixMulKernel(float *A, float *B, float *C, int width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ float As[16][16];
    __shared__ float Bs[16][16];

    for (int k = 0; k < width / 16; ++k) {
        As[threadIdx.y][threadIdx.x] = A[row * width + k * 16 + threadIdx.x];
        Bs[threadIdx.y][threadIdx.x] = B[(k * 16 + threadIdx.y) * width + col];
        __syncthreads();
        float value = 0.0f;
        for (int n = 0; n < 16; ++n) {
            value += As[threadIdx.y][n] * Bs[n][threadIdx.x];
        }
        C[row * width + col] += value;
    }
}